#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cutil.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <algorithm>
#include <assert.h>
#include "cbcl_model.h"

const int BLOCK_SIZE=16; 
const int MAXTHREADS =128; 

using namespace std;


__global__  void kernel_c_generic(band_info* d_outbands,int b,float scalex,float scaley,int pool_xy,int blockrows);
__global__  void kernel_s_exp_tuning(band_info* filters,band_info* s,int band,int blockrows);

/*
image texture
*/
texture<float,2,hipReadModeElementType> teximg;
texture<float,2,hipReadModeElementType> texfilt;

__host__ __device__ float* elptr(float* base,int depth,int row,int col,int height,int pitch)
{
	return (float*)((char*)base+depth*height*pitch+row*pitch)+col;
}

void gpu_to_cpu(band_info* pcin,int num_bands,band_info** ppcout,int copy)
{
	*ppcout				= new band_info[num_bands]; /*final*/
	band_info* hband	= new band_info[num_bands]; /*staging*/
	/*copy structure*/
	CUDA_SAFE_CALL(hipMemcpy(hband,pcin,sizeof(band_info)*num_bands,hipMemcpyDeviceToHost));
	assert(*ppcout);
	assert(hband);

	for(int i=0;i<num_bands;i++)
	{
		band_info* pcout	=	*ppcout+i;	
		float*     cpuptr	=	NULL;
		pcout->depth		=	hband[i].depth;
		pcout->height		=	hband[i].height;
		pcout->width		=	hband[i].width;										
        pcout->where        =   ONHOST;
		/*copy*/
		if(copy)
		{
			/*allocate memory*/
			cpuptr		 = new float[hband[i].height*hband[i].width*hband[i].depth];
			CUDA_SAFE_CALL(hipMemcpy2D(cpuptr,hband[i].width*sizeof(float),
						 hband[i].ptr,hband[i].pitch,
						 hband[i].width*sizeof(float),hband[i].height*hband[i].depth,
						 hipMemcpyDeviceToHost));
			pcout->ptr   = cpuptr;
			pcout->pitch = hband[i].width*sizeof(float);
		}
		else
		{
			pcout->ptr	 = NULL;
			pcout->pitch = 0;
		}
		
	}
	delete[] hband;
}


void cpu_to_gpu(band_info* pcin,int num_bands,band_info** ppcout,int copy)
{
	band_info* hband = new band_info[num_bands];
	/*stage the structure in the host*/
	for(int i=0;i<num_bands;i++)
	{
		band_info* pcout	= hband+i;	
		float*     gpuptr	= NULL;
		size_t     pitch	= 0;
		pcout->depth		= pcin[i].depth;
		pcout->height		= pcin[i].height;
		pcout->width		= pcin[i].width;										
        pcout->where        = ONDEVICE;
		if(copy)
		{
			/*allocate space*/
			CUDA_SAFE_CALL(hipMallocPitch((void**)&gpuptr,&pitch,
											pcin[i].width*sizeof(float),
											pcin[i].height*pcin[i].depth));
			/*copy*/
			CUDA_SAFE_CALL(hipMemcpy2D(gpuptr,pitch,
										pcin[i].ptr,pcin[i].pitch,
										pcin[i].width*sizeof(float),pcin[i].height*pcin[i].depth,
										hipMemcpyHostToDevice));
			pcout->ptr   = gpuptr;
			pcout->pitch = pitch;
		}
		else
		{
			pcout->ptr   = NULL; /*NEVER USE THIS*/
			pcout->pitch = 0;
		}
	}
	/*copy onto the gpu*/
	CUDA_SAFE_CALL(hipMalloc((void**)ppcout,num_bands*sizeof(band_info)));
	CUDA_SAFE_CALL(hipMemcpy(*ppcout,hband,num_bands*sizeof(band_info),hipMemcpyHostToDevice));
	delete[] hband;
}

void gpu_release_images(band_info** ppbands,int num_bands)
{
	band_info* hbands = new band_info[num_bands]; /*staging*/
	assert(hbands);
	/*copy structure*/
	CUDA_SAFE_CALL(hipMemcpy(hbands,*ppbands,sizeof(band_info)*num_bands,hipMemcpyDeviceToHost));
	assert(*ppbands);
	for(int i=0;i<num_bands;i++)
	{
		band_info* pband = hbands+i;	
		void* ptr		 = pband->ptr;
		if(ptr)	hipFree(ptr);
	}
	delete[] hbands;
	hipFree(*ppbands);
}


__global__ void kernel_c_generic(band_info* d_outbands,int b,float dx,float dy,int pool_xy,int blockrows)
{
	int depth     = blockIdx.x;
	int c_depth   = d_outbands[b].depth;
	int c_height  = d_outbands[b].height;
	int c_width	  = d_outbands[b].width;
	int c_pitch   = d_outbands[b].pitch;
	int row_start = threadIdx.x*blockrows;
	int row_end   = row_start+blockrows;
	int	bound     = pool_xy/2;
	float pixval  = 0;

	for(int row=row_start;row<row_end && row<c_height;row++)
	{
		float    cy   =  (float)(row+c_height*depth)/(c_height*c_depth);
		if(row< bound || row>= c_height-bound) continue;
		for(int col=bound;col< c_width-bound;col++)
		{
			float* outptr = elptr(d_outbands[b].ptr,depth,row,col,c_height,c_pitch);
			float  maxval = *outptr;
			/*get maximum*/
			float    cx   =  (float)col/c_width;
			for(int u=-bound;u<=bound;u++)
			{
				for(int v=-bound;v<=bound;v++)
				{
					pixval		 = tex2D(teximg,cx+u*dx,cy+v*dy);
					maxval       = fmaxf(maxval,pixval);
				}/*end v*/
			}/*end u*/
			*outptr= pixval;
		}/*end col*/
	}/*end row*/
}

void gpu_c_local(
		IN  band_info* sin,     /*pointer to DEVICE storage*/
		IN  int in_bands,     /*number of input bands*/
		IN  int pool_xy,      /*spatial pooling: subsampling by pool_xy/2*/
		IN  int pool_scale,   /*scale wise pooling: out_bands=in_bands/pool_scale*/
		OUT band_info** ppc,      /*pointer to DEVICE storage*/
		OUT int	*pout_bands   /*number of output bands*/
	)
{
   hipArray*				gpu_img_array;
   band_info*				d_outbands;
   band_info*				h_outbands;
   float*					d_ptr;
   size_t					d_pitch;
   int i,o,b;

   int out_bands = in_bands/pool_scale;
   *pout_bands   = out_bands;

   /*stage output*/
   h_outbands = new band_info[out_bands];
   int srate  = pool_xy/2; 
   for(i=0,o=0;i<in_bands;i+=pool_scale,o++)
   {
		h_outbands[o].height = sin[i].height/srate;
		h_outbands[o].width  = sin[i].width/srate;
		h_outbands[o].depth  = sin[i].depth;
		CUDA_SAFE_CALL(hipMallocPitch((void**)&d_ptr,&d_pitch,h_outbands[o].width*sizeof(float),h_outbands[o].depth*h_outbands[o].height));
		CUDA_SAFE_CALL(hipMemset2D(d_ptr,d_pitch,0,h_outbands[o].width*sizeof(float),h_outbands[o].depth*h_outbands[o].height));
		h_outbands[o].pitch = d_pitch;
		h_outbands[o].ptr   = d_ptr;
   }
   CUDA_SAFE_CALL(hipMalloc((void**)&d_outbands,out_bands*sizeof(band_info)));
   CUDA_SAFE_CALL(hipMemcpy(d_outbands,h_outbands,out_bands*sizeof(band_info),hipMemcpyHostToDevice));

  
   /*copy image*/ 
   for(b=0;b<in_bands;b++)
   {
	   hipChannelFormatDesc	imgdesc=hipCreateChannelDesc<float>();
	   CUDA_SAFE_CALL(hipMallocArray(&gpu_img_array,&imgdesc,sin[b].width,sin[b].height*sin[b].depth));
		/*bind the texture*/
		teximg.addressMode[0] = hipAddressModeClamp;
	    teximg.addressMode[1] = hipAddressModeClamp;
	    teximg.filterMode     = hipFilterModePoint; //take note//
	    teximg.normalized     = true;//take note//
		/*copy to array*/
		CUDA_SAFE_CALL(hipMemcpy2DToArray(gpu_img_array,0,0,
										   sin[b].ptr,sin[b].pitch,
										   sin[b].width*sizeof(float),sin[b].height*sin[b].depth,
									       hipMemcpyHostToDevice));
	    CUDA_SAFE_CALL(hipBindTextureToArray(teximg,gpu_img_array));
		
		/*call the kernel*/
		o				 = b/pool_scale;
		uint3 gridsz	 = make_uint3(sin[b].depth,1,1);
		int   nthreads	 = min(h_outbands[o].height,MAXTHREADS);
		int   blockrows  = ceilf((float)h_outbands[o].height/nthreads);
		uint3 blocksz	 = make_uint3(nthreads,1,1);
		float dx	     = 1.0f/sin[b].width;
		float dy		 = 1.0f/(sin[b].height*sin[b].depth);
		kernel_c_generic<<<gridsz,blocksz>>>(d_outbands,o,dx,dy,pool_xy,blockrows);
		CUDA_SAFE_CALL(hipDeviceSynchronize());
		CUDA_SAFE_CALL(hipUnbindTexture(teximg));						   
		CUDA_SAFE_CALL(hipFreeArray(gpu_img_array));
   }
   
   /*copy image to output*/   
   gpu_to_cpu(d_outbands,out_bands,ppc);
   /*clean up*/
   delete [] h_outbands;
   gpu_release_images(&d_outbands,out_bands);
}


void cpu_release_images(band_info** ppbands,int num_bands)
{
	for(int i=0;i<num_bands;i++)
	{
		delete[] (*ppbands)[i].ptr;
	}
	delete [] *ppbands;
	*ppbands = NULL;
}

__global__ void kernel_resize(float *dest,int wt,int ht,const int TILESZ)
{
    int row=blockIdx.y*TILESZ+threadIdx.y;
    int col=blockIdx.x*TILESZ+threadIdx.x;
    if(row>=ht) return;
    if(col>=wt) return;
    dest[row*wt+col]=tex2D(teximg,(float)col/wt,(float)row/ht);
}

void gpu_create_c0(float* pimg,int width,int height,band_info** ppc,int* pbands,float scale,int num_scales)
{
	*ppc				   = new band_info[num_scales];
	*pbands				   = num_scales;
    assert(*ppc!=NULL);
    assert(*pbands>=1);
    /*create first band*/
    (*ppc)->height           = height;
    (*ppc)->width            = width;
    (*ppc)->depth            = 1;
    (*ppc)->pitch            = width*sizeof(float);
    (*ppc)->ptr              = new float[height*width];
    assert((*ppc)->ptr);
    memcpy((*ppc)->ptr,pimg,height*width*sizeof(float));

    hipArray*               pdimg;
    hipChannelFormatDesc	imgdesc=hipCreateChannelDesc<float>();
    CUDA_SAFE_CALL(hipMallocArray(&pdimg,&imgdesc,width,height));
	/*bind the texture*/
	teximg.addressMode[0] = hipAddressModeWrap;
	teximg.addressMode[1] = hipAddressModeWrap;
	teximg.filterMode     = hipFilterModeLinear;
	teximg.normalized     = true;
	/*copy to array*/
	CUDA_SAFE_CALL(hipMemcpy2DToArray(pdimg,0,0,
										   pimg,width*sizeof(float),
										   width*sizeof(float),height,
									       hipMemcpyHostToDevice));
	CUDA_SAFE_CALL(hipBindTextureToArray(teximg,pdimg));

    float* pdout             = NULL;
    CUDA_SAFE_CALL(hipMalloc((void**)&pdout,height*width*sizeof(float)));

	for(int b=1,curr_scale=scale;b<num_scales;b++)
	{
        band_info* prev          = *ppc+b-1;
        band_info* pc            = *ppc+b;
		int bht			         = roundf(prev->height/scale);
		int bwt			         = roundf(prev->width/scale);
		pc->height		         = bht;
		pc->width		         = bwt;
		pc->pitch		         = bwt*sizeof(float);
		pc->depth		         = 1;
		pc->ptr			         = new float[bht*bwt];
	    /*call the kernel*/
        const int TILESZ = 8;
		uint3 gridsz	 = make_uint3(ceilf(bwt/TILESZ),ceilf(bht/TILESZ),1);
		uint3 blocksz	 = make_uint3(TILESZ,TILESZ,1);
		kernel_resize<<<gridsz,blocksz>>>(pdout,bwt,bht,TILESZ);
        CUDA_SAFE_CALL(hipMemcpy(pc->ptr,pdout,bht*bwt*sizeof(float),hipMemcpyDeviceToHost));
        CUDA_SAFE_CALL(hipDeviceSynchronize());
   }
   CUDA_SAFE_CALL(hipFree(pdout));
   CUDA_SAFE_CALL(hipUnbindTexture(teximg));						   
}

void cpu_create_c0(float* pimg,int width,int height,band_info** ppc,int* pbands,float scale,int num_scales)
{
	*ppc				   = new band_info[num_scales];
	*pbands				   = num_scales;
	assert(*ppc!=NULL);
    assert(*pbands>=1);
    /*create first band*/
    band_info*  prev         = *ppc;
    prev->height           = height;
    prev->width            = width;
    prev->depth            = 1;
    prev->pitch            = width*sizeof(float);
    prev->ptr              = new float[height*width];
    memcpy(prev->ptr,pimg,height*width*sizeof(float));
    assert(prev->ptr);
    /*determine dynamic range*/
    float minval           = *min_element(pimg,&pimg[height*width]);
    float maxval           = *max_element(pimg,&pimg[height*width]);
    float range            = maxval-minval+1e-6;

    /*create the other bands recursively*/
	for(int b=1;b<num_scales;b++,prev++)
	{
		pimg            = prev->ptr;
        width           = prev->width;
        height          = prev->height;
        
        band_info* pc	= *ppc+b;
		int bht			= roundf(height/scale);
		int bwt			= roundf(width/scale);
		pc->height		= bht;
		pc->width		= bwt;
		pc->pitch		= bwt*sizeof(float);
		pc->depth		= 1;
		pc->ptr			= new float[bht*bwt];

		assert(pc->ptr!=NULL);
        float cmin      = 1e6; /*current min*/
        float cmax      = 0;   /*current max*/
		for(int x=0;x<bwt;x++)
		{
			for(int y=0;y<bht;y++)
			{
				float sx = x*scale;
				float sy = y*scale;
				int   fx = floorf(sx); int  cx = ceilf(sx);cx=(cx>=width)?(width-1):cx;
				int   fy = floorf(sy); int  cy = ceilf(sy);cy=(cy>=height)?(height-1):cy;
				float xalpha=sx-fx;
				float yalpha=sy-fy;
				float val   =pimg[fx+fy*width]*(1-xalpha)*(1-yalpha)+
							 pimg[cx+fy*width]*(xalpha)*(1-yalpha)+
							 pimg[fx+cy*width]*(1-xalpha)*(yalpha)+
							 pimg[cx+cy*width]*(xalpha)*(yalpha);
				pc->ptr[y*bwt+x]=val;
                if(val<cmin) cmin=val;
                if(val>cmax) cmax=val;
			}
		}
        float crange = cmax-cmin+1e-6; 
        float factor = range/crange;
        for(int i=0;i<bht*bwt;i++)
            pc->ptr[i]=(pc->ptr[i]-cmin)*factor+minval;
	}
}


__global__  void kernel_s_norm_filter(float* dest,int pitch,int depth,int wt,int ht,int fwt,int fht)
{
    int         row             = blockIdx.y*BLOCK_SIZE+threadIdx.y;
    int         col             = blockIdx.x*BLOCK_SIZE+threadIdx.x;
    int         xoff            = floorf(fwt/2);
    int         yoff            = floorf(fht/2);
	int u,v,d;
    float       den             = 0;
    float       num             = 0;
    float       pixval          = 0;
    float       filtval         = 0;
    if(row>ht-fht) return;
    if(col>wt-fwt) return;
    for(d=0;d<depth;d++)
    {
        for(v=0;v<fht;v++)
            for(u=0;u<fwt;u++)
            {
                pixval =tex2D(teximg,col+u,d*ht+row+v);
                filtval=tex2D(texfilt,u,d*fht+v);
                num    +=pixval*filtval;
                den    +=pixval*pixval;
            }
    }
    /*printf(".");*/
    *elptr(dest,0,row+yoff,col+xoff,ht,pitch)=fabs(num)/sqrtf(den+1e-6);
}

/*
put the image into texture memory
put the filter into global memory
call the kernel for each band of the input (maybe change later)
*/
void gpu_s_norm_filter(band_info* cin,int in_bands,band_info* filt,int num_filt, band_info** pps, int *out_bands)
{
   hipArray*				imgarray;
   hipArray*               filtarray;
   band_info*				h_outbands;
   float*					d_ptr;
   size_t					d_pitch;
   /*channel description*/
   
   /*stage output*/
   h_outbands = new band_info[in_bands];
   for(int b=0;b<in_bands;b++)
   {
		h_outbands[b].height = cin[b].height;
		h_outbands[b].width  = cin[b].width;
		h_outbands[b].depth  = num_filt;
		CUDA_SAFE_CALL(hipMalloc((void**)&d_ptr,cin[b].width*sizeof(float)*num_filt*cin[b].height));
		CUDA_SAFE_CALL(hipMemset(d_ptr,0,cin[b].width*sizeof(float)*num_filt*cin[b].height));
		h_outbands[b].pitch = cin[b].width*sizeof(float);
		h_outbands[b].ptr   = d_ptr;
   }
   *pps      = h_outbands;
   *out_bands= in_bands;
	   
   /*copy image*/ 
   hipChannelFormatDesc	imgdesc=hipCreateChannelDesc<float>();
   hipChannelFormatDesc    filtdesc=hipCreateChannelDesc<float>();
   CUDA_SAFE_CALL(hipMallocArray(&filtarray,&filtdesc,filt[0].width,filt[0].height*filt[0].depth));
   CUDA_SAFE_CALL(hipMallocArray(&imgarray,&imgdesc,cin[0].width,cin[0].height*cin[0].depth));
   /*fix address modes*/
    teximg.addressMode[0] = hipAddressModeClamp;
    teximg.addressMode[1] = hipAddressModeClamp;
    teximg.filterMode     = hipFilterModePoint;
    teximg.normalized     = false;
    
    texfilt.addressMode[0] = hipAddressModeClamp;
    texfilt.addressMode[1] = hipAddressModeClamp;
    texfilt.filterMode     = hipFilterModePoint;
    texfilt.normalized     = false;

    /*call the kernel*/
   for(int b=0;b<in_bands;b++)
   {
	/*copy to array*/
		CUDA_SAFE_CALL(hipMemcpy2DToArray(imgarray,0,0,
										   cin[b].ptr,cin[b].pitch,
										   cin[b].width*sizeof(float),cin[b].height*cin[b].depth,
									       hipMemcpyHostToDevice));
	    CUDA_SAFE_CALL(hipBindTextureToArray(teximg,imgarray));
        for(int f=0;f<num_filt;f++)
        {
            CUDA_SAFE_CALL(hipMemcpy2DToArray(filtarray,0,0,
                                               filt[f].ptr,filt[f].width*sizeof(float),
                                               filt[f].width*sizeof(float),filt[f].height*filt[f].depth,
                                               hipMemcpyHostToDevice));
	        CUDA_SAFE_CALL(hipBindTextureToArray(texfilt,filtarray));
		    uint3 gridsz	 = make_uint3(ceilf((float)cin[b].width/BLOCK_SIZE),ceilf((float)cin[b].height/BLOCK_SIZE),1);
		    uint3 blocksz	 = make_uint3(BLOCK_SIZE,BLOCK_SIZE,1);
            float* dest      = elptr(h_outbands[b].ptr,f,0,0,h_outbands[b].height,h_outbands[b].pitch);
		    kernel_s_norm_filter<<<gridsz,blocksz>>>(dest,h_outbands[b].pitch,cin[b].depth,cin[b].width,cin[b].height,filt[f].width,filt[f].height);
            CUDA_SAFE_CALL(hipDeviceSynchronize());
	        CUDA_SAFE_CALL(hipUnbindTexture(texfilt));
        }
        CUDA_SAFE_CALL(hipUnbindTexture(teximg));
   }
   for(int b=0;b<in_bands;b++)
   {
       int    sz  = h_outbands[b].height*h_outbands[b].width*num_filt;
       float* ptr = new float[sz];
       assert(ptr!=NULL);
       CUDA_SAFE_CALL(hipMemcpy(ptr,h_outbands[b].ptr,sz*sizeof(float),hipMemcpyDeviceToHost));
       CUDA_SAFE_CALL(hipFree(h_outbands[b].ptr));
       h_outbands[b].ptr   =ptr;
       h_outbands[b].pitch =h_outbands[b].width*sizeof(float);
   }
   CUDA_SAFE_CALL(hipDeviceSynchronize());
   /*copy image to output*/   
   CUDA_SAFE_CALL(hipFreeArray(imgarray));
   CUDA_SAFE_CALL(hipFreeArray(filtarray));
}


void gpu_s_rbf(band_info* cin,int in_bands,band_info* filt,int num_filt,OUT band_info** pps,int*out_bands)
{
   hipArray*				gpu_img_array;
   band_info*				d_outbands;
   band_info*				h_outbands;
   band_info*				d_filts;
   float*					d_ptr;
   size_t					d_pitch;
   /*channel description*/
   
   /*stage output*/
   h_outbands = new band_info[in_bands];
   for(int b=0;b<in_bands;b++)
   {
		h_outbands[b].height = cin[b].height;
		h_outbands[b].width  = cin[b].width;
		h_outbands[b].depth  = num_filt;
		CUDA_SAFE_CALL(hipMallocPitch((void**)&d_ptr,&d_pitch,cin[b].width*sizeof(float),num_filt*cin[b].height));
		CUDA_SAFE_CALL(hipMemset2D(d_ptr,d_pitch,0,cin[b].width*sizeof(float),num_filt*cin[b].height));
		h_outbands[b].pitch = d_pitch;
		h_outbands[b].ptr   = d_ptr;
   }
   CUDA_SAFE_CALL(hipMalloc((void**)&d_outbands,in_bands*sizeof(band_info)));
   CUDA_SAFE_CALL(hipMemcpy(d_outbands,h_outbands,in_bands*sizeof(band_info),hipMemcpyHostToDevice));
   *out_bands= in_bands;
	   
   /* transfer filters*/
   cpu_to_gpu(filt,num_filt,&d_filts);
  
   /*copy image*/ 
   hipChannelFormatDesc	imgdesc=hipCreateChannelDesc<float>();
   CUDA_SAFE_CALL(hipMallocArray(&gpu_img_array,&imgdesc,cin[0].width,cin[0].height*cin[0].depth));
   for(int b=0;b<in_bands;b++)
   {
		/*bind the texture*/
		teximg.addressMode[0] = hipAddressModeClamp;
	    teximg.addressMode[1] = hipAddressModeClamp;
	    teximg.filterMode     = hipFilterModePoint;
	    teximg.normalized     = false;
		/*copy to array*/
		CUDA_SAFE_CALL(hipMemcpy2DToArray(gpu_img_array,0,0,
										   cin[b].ptr,cin[b].pitch,
										   cin[b].width*sizeof(float),cin[b].height*cin[b].depth,
									       hipMemcpyHostToDevice));
	    CUDA_SAFE_CALL(hipBindTextureToArray(teximg,gpu_img_array));
		/*call the kernel*/
		int   nthreads	 = min(cin[b].height,MAXTHREADS);
		int   blockrows  = ceilf((float)cin[b].height/nthreads);
		uint3 gridsz	 = make_uint3(num_filt,1,1);
		uint3 blocksz	 = make_uint3(nthreads,1,1);
		kernel_s_exp_tuning<<<gridsz,blocksz>>>(d_filts,d_outbands,b,blockrows);
	    CUDA_SAFE_CALL(hipUnbindTexture(teximg));						   
   }
   CUDA_SAFE_CALL(hipDeviceSynchronize());
   /*copy image to output*/   
   gpu_to_cpu(d_outbands,*out_bands,pps);
   /*clean up*/
   delete [] h_outbands;
   CUDA_SAFE_CALL(hipFreeArray(gpu_img_array));
   gpu_release_images(&d_outbands,in_bands);
   gpu_release_images(&d_filts,num_filt);
}


__global__ void kernel_s_exp_tuning(band_info* filters,band_info* s,int band,int blockrows)
{
	__shared__ float sfilt[1024];
	/*load the filter into shared memory*/
	band_info 	filt_curr		=filters[blockIdx.x];
	int			filt_pitch		=filt_curr.pitch;
	int			filt_width		=filt_curr.width;
	int			filt_height		=filt_curr.height;
	int			filt_depth		=filt_curr.depth;
	int			s_height		=s[band].height;
	int			s_width			=s[band].width;
	int			s_pitch			=s[band].pitch;

	float		*inptr,*outptr;
	int			depth			= 0;
	int			col				= 0;
	int			row_start		=threadIdx.x*blockrows;
	int			row_end			=row_start+blockrows;
    int			row				=row_start;
	int u,v;

	for(row = row_start;row<filt_height && row<row_end;row++)
	{
		for(depth=0;depth<filt_depth;depth++)
		{
			for(col=0;col<filt_width;col++)
			{
				inptr  		= elptr(filt_curr.ptr,depth,row,col,filt_height,filt_pitch);
				outptr		= elptr(sfilt,depth,row,col,filt_height,filt_width*sizeof(float));
				*outptr		= *inptr;
			}	
		}
	}
	__syncthreads();
	int  bound = filt_width/2;
	for(row = row_start;row< s_height&& row<row_end;row++)
	{
		/*compute response for a single row of output*/
		if(row<bound || row>= s_height-bound)
			continue;
		outptr			=	elptr(s[band].ptr,blockIdx.x,row,0,s_height,s_pitch);
		for(col=0;col<bound;col++)
			outptr[col]=0;
		for(col=bound;col<s_width-bound;col++)
		{
			float num      = 0.0f;
			float den      = 0.01f;
			for(depth=0;depth<1;depth++)
			{
				for(u=0;u<filt_width;u++)
				{
					for(v=0;v<filt_height;v++)
					{
						float  pixval  = tex2D(teximg,col+u-bound,s_height*depth+row+v-bound);
						float* pfiltval= elptr(sfilt,depth,v,u,filt_height,filt_width*sizeof(float));
						num+=  ((*pfiltval)-pixval)*((*pfiltval)-pixval);
						den+= (*pfiltval)*(*pfiltval);
					}/*end y*/
				}/*end x*/
			}/*end depth*/
			float sigma = sqrtf(den)*0.33;
			sigma       = 2*sigma*sigma;
			float outval= expf(-num/sigma);
			outptr[col] = outval;//fminf(1,outval);
		}/*end col*/
		for(col=s_width-bound;col<s_width;col++)
			outptr[col]=0;
	}/*end row*/
	__syncthreads();
}

void cpu_c_global(
	IN band_info* s,      /*pointer to device storage*/
	IN int in_bands,      /*number of input bands*/
	OUT float** ppc,          /*pointer to DEVICE storage*/
	OUT int* out_units   /*=input depth*/	
)
{
	*out_units = s[0].depth;
	*ppc       = new float[*out_units];
	assert(*ppc);
	
	float* pc  = *ppc;
	memset(pc,0,sizeof(float)*(*out_units));

	for(int d=0;d<s[0].depth;d++)
	{
		for(int b=0;b<in_bands;b++)
		{
			int    numel  = s[b].height*s[b].width;
			float* ptr    = s[b].ptr+d*numel;
			float* pmaxval= max_element(ptr,ptr+numel);
			pc[d]         = max(*pmaxval,pc[d]);
		}
	}
}


void callback_c1_baseline(band_info* cin,int ncin, band_info* filts,int nfilts,band_info** ppcout,int* pncout)
{
	band_info* sout;
	int        nsout;
	gpu_s_norm_filter(cin,ncin,filts,nfilts,&sout,&nsout);
	//gpu_s_norm_filter(cin,ncin,filts,nfilts,ppcout,pncout);
	gpu_c_local(sout,nsout,8,2,ppcout,pncout);
	cpu_release_images(&sout,nsout);
}

void callback_c2_baseline(band_info* cin,int ncin,
						  band_info* c0filts,int nc0filts,
						  band_info* c1filts,int nc1filts,
						  band_info** ppcout,int* pncout)
{
	band_info	*s1,*c1,*s2;
	int         ns1,nc1,ns2;
	gpu_s_norm_filter(cin,ncin,c0filts,nc0filts,&s1,&ns1);
	gpu_c_local(s1,ns1,8,2,&c1,&nc1);
	gpu_s_rbf(c1,nc1,c1filts,nc1filts,&s2,&ns2);
	gpu_c_local(s2,ns2,5,2,ppcout,pncout);

	cpu_release_images(&s1,ns1);
	cpu_release_images(&c1,nc1);
	cpu_release_images(&s2,ns2);
}

void callback_c2b_baseline(band_info* cin,int ncin,
						  band_info* c0filts,int nc0filts,
						  band_info* c1filts,int nc1filts,
						  float** ppc2b,int* nc2b)
{
	band_info	*s1,*c1,*s2;
	int         ns1,nc1,ns2;
	gpu_s_norm_filter(cin,ncin,c0filts,nc0filts,&s1,&ns1);
	gpu_c_local(s1,ns1,8,2,&c1,&nc1);
	gpu_s_rbf(c1,nc1,c1filts,nc1filts,&s2,&ns2);
	cpu_c_global(s2,ns2,ppc2b,nc2b);
	
	//cpu_c_global(c1,nc1,ppc2b,nc2b);
	cpu_release_images(&s1,ns1);
	cpu_release_images(&c1,nc1);
	cpu_release_images(&s2,ns2);
}
